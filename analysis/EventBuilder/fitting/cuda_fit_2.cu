#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "timer.h"

#define N 1024     // Number of data points
#define THREADS 256
#define BLOCKS (N + THREADS - 1) / THREADS

float model(float x, float A, float T, float R, float B) {
  return A / (1.0f + expf((x - T) / R)) + B;
}

__global__ void compute_residuals_SSR_and_jacobian(float* x, float* y, float* residuals, float *SSR, float* J, float A, float T, float R, float B) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N) return;

  float xi = x[i];
  float yi = y[i];
  float exp_term = expf((xi - T) / R);
  float denom = (1.0f + exp_term);
  float denom2 = denom * denom;

  float fi = A / denom + B;
  float ri = yi - fi;
  residuals[i] = ri;
  atomicAdd(SSR, ri * ri);

  // Compute partial derivatives (Jacobian)
  J[i * 4 + 0] = -1.0f / denom;                        // ∂f/∂A
  J[i * 4 + 1] = (A * exp_term) / (R * denom2);        // ∂f/∂T
  J[i * 4 + 2] = (A * exp_term * (xi - T)) / (R * R * denom2); // ∂f/∂R
  J[i * 4 + 3] = -1.0f;                                // ∂f/∂B
  
}

// Reduction kernel to compute JᵗJ and Jᵗr
__global__ void compute_JTJ_JTr(float* J, float* residuals, float* JTJ, float* JTr) {
  __shared__ float sJTJ[4 * 4];
  __shared__ float sJTr[4];

  int tid = threadIdx.x;
  for (int i = tid; i < 4 * 4; i += blockDim.x) sJTJ[i] = 0.0f;
  for (int i = tid; i < 4; i += blockDim.x) sJTr[i] = 0.0f;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
      float* Ji = &J[i * 4];
      float ri = residuals[i];
      for (int j = 0; j < 4; ++j) {
          atomicAdd(&sJTr[j], Ji[j] * ri);
          for (int k = 0; k < 4; ++k) {
              atomicAdd(&sJTJ[j * 4 + k], Ji[j] * Ji[k]);
          }
      }
  }
  __syncthreads();

  if (tid < 4 * 4) atomicAdd(&JTJ[tid], sJTJ[tid]);
    if (tid < 4)     atomicAdd(&JTr[tid], sJTr[tid]);
}


// Host-side LMA update (solve (JᵗJ + λI) δ = Jᵗr)
float update_parameters(float* JTJ, float* JTr, float* params, float lambda) {
  // Solve 4x4 linear system using basic Gaussian elimination (naive, for demo)
  float A[4][5]; // 4x4 + rhs
  for (int i = 0; i < 4; ++i)
      for (int j = 0; j < 4; ++j)
          A[i][j] = JTJ[i * 4 + j] + (i == j ? lambda : 0.0f);
  for (int i = 0; i < 4; ++i)
      A[i][4] = JTr[i];

  // Gaussian elimination
  for (int i = 0; i < 4; ++i) {
      float pivot = A[i][i];
      for (int j = 0; j <= 4; ++j)
          A[i][j] /= pivot;
      for (int k = 0; k < 4; ++k) {
          if (k == i) continue;
          float factor = A[k][i];
          for (int j = 0; j <= 4; ++j)
              A[k][j] -= factor * A[i][j];
      }
  }

  // Update parameters
  for (int i = 0; i < 4; ++i) params[i] += A[i][4];

  // calculate SSR
  float SSR = 0.0f;
  for (int i = 0; i < N; ++i) {
      float fi = model(i * 0.01f, params[0], params[1], params[2], params[3]);
      float ri = (i * 0.01f - fi);
      SSR += ri * ri;
  }
  return SSR;
    
}

int main() {
    float *x, *y, *d_x, *d_y, *d_res, *d_J, *d_JTJ, *d_JTr, *d_SSR;
    float params[4] = { 5.0f, 5.0f, 1.0f, 0.5f }; // Initial guess: A, T, R, B

    x = new float[N];
    y = new float[N];

    // Generate synthetic data
    for (int i = 0; i < N; ++i) {
        x[i] = i * 0.01f;
        float true_val = 3.0f / (1.0f + expf((x[i] - 7.0f) / 0.8f)) + 1.0f;
        y[i] = true_val + 0.05f * ((rand() % 1000) / 1000.0f - 0.5f);
    }

    unsigned int time0 = getTime_us();

    // Allocate GPU memory
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_res, N * sizeof(float));
    hipMalloc(&d_SSR, sizeof(float));
    hipMalloc(&d_J, N * 4 * sizeof(float));
    hipMalloc(&d_JTJ, 16 * sizeof(float));
    hipMalloc(&d_JTr, 4 * sizeof(float));

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    float lambda = 1e-3f;

    unsigned int time1 = getTime_us();
    printf("Memory allocation and data transfer took %u us\n", time1 - time0 );

    for (int iter = 0; iter < 20; ++iter) {

      unsigned int time2 = getTime_us();

      compute_residuals_SSR_and_jacobian<<<BLOCKS, THREADS>>>(d_x, d_y, d_res, d_SSR, d_J, params[0], params[1], params[2], params[3]);
      compute_JTJ_JTr<<<BLOCKS, THREADS>>>(d_J, d_res, d_JTJ, d_JTr);

      hipDeviceSynchronize();

      unsigned int time2a = getTime_us();
      printf("Iteration %d: Kernel execution took %u us\n", iter, time2a - time2);

      float JTJ[16], JTr[4], SSR;
      hipMemcpy(JTJ, d_JTJ, sizeof(float) * 16, hipMemcpyDeviceToHost);
      hipMemcpy(JTr, d_JTr, sizeof(float) * 4, hipMemcpyDeviceToHost);
      hipMemcpy(&SSR, d_SSR, sizeof(float), hipMemcpyDeviceToHost);

      unsigned int time2b = getTime_us();
      printf("Iteration %d: Data transfer took %u us\n", iter, time2b - time2a);

      float new_SSR = update_parameters(JTJ, JTr, params, lambda);
      unsigned int time3 = getTime_us();
      printf("Iteration %d: Time for kernel execution and parameter update: %u us\n", iter, time3 - time2);

      printf("Iteration %d: A = %.4f, T = %.4f, R = %.4f, B = %.4f | SSR: %f, new SSR: %f\n",
             iter, params[0], params[1], params[2], params[3], SSR, new_SSR);

    }

    printf("Fitted parameters:\nA = %.4f\nT = %.4f\nR = %.4f\nB = %.4f\n",
        params[0], params[1], params[2], params[3]);

    unsigned int time4 = getTime_us();
    printf("Total execution time: %u us\n", time4 - time0);

    // Cleanup
    hipFree(d_x); hipFree(d_y); hipFree(d_res); hipFree(d_J);
    hipFree(d_JTJ); hipFree(d_JTr);
    delete[] x; delete[] y;

    return 0;
}
