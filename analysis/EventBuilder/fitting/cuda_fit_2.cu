#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "timer.h"

#define N 1000     // Number of data points
#define P 4       // Number of parameters (A, T, R, B)
#define dF (N - P) // Degrees of freedom
#define THREADS 256
#define BLOCKS (N + THREADS - 1) / THREADS

float model(float x, float A, float T, float R, float B) {
  return A / (1.0f + expf((x - T) / R)) + B;
}

__global__ void compute_residuals_SSR_and_jacobian(float* y, float* residuals, float *SSR, float* J, float A, float T, float R, float B) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N) return;

  float yi = y[i];
  float exp_term = expf((i - T) / R);

  if( exp_term > 1e20f ){

    float ri = yi - B;
    residuals[i] = ri;
    atomicAdd(SSR, ri * ri);

    // Compute partial derivatives (Jacobian)
    J[i * P + 0] = 0;                        // ∂f/∂A
    J[i * P + 1] = 0;        // ∂f/∂T
    J[i * P + 2] = 0; // ∂f/∂R
    J[i * P + 3] = 1.0f;                                // ∂f/∂B

  }else{
    float denom = (1.0f + exp_term);
    float denom2 = denom * denom;

    float fi = A / denom + B;
    float ri = yi - fi;
    residuals[i] = ri;
    atomicAdd(SSR, ri * ri);

    // Compute partial derivatives (Jacobian)
    J[i * P + 0] = 1.0f / denom;                        // ∂f/∂A
    J[i * P + 1] = (A * exp_term) / (R * denom2);        // ∂f/∂T
    J[i * P + 2] = (A * exp_term * (i - T)) / (R * R * denom2); // ∂f/∂R
    J[i * P + 3] = 1.0f;                                // ∂f/∂B

  }
}

// Reduction kernel to compute JᵗJ and Jᵗr
__global__ void compute_JTJ_JTr(float* J, float* residuals, float* JTJ, float* JTr) {
  __shared__ float sJTJ[P * P];
  __shared__ float sJTr[P];

  int tid = threadIdx.x;
  for (int i = tid; i < P * P; i += blockDim.x) sJTJ[i] = 0.0f;
  for (int i = tid; i < P; i += blockDim.x) sJTr[i] = 0.0f;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
      float* Ji = &J[i * P];
      float ri = residuals[i];
      for (int j = 0; j < P; ++j) {
          atomicAdd(&sJTr[j], Ji[j] * ri);
          for (int k = 0; k < P; ++k) {
              atomicAdd(&sJTJ[j * P + k], Ji[j] * Ji[k]);
          }
      }
  }
  __syncthreads();

  if (tid < P * P) atomicAdd(&JTJ[tid], sJTJ[tid]);
  if (tid < P)     atomicAdd(&JTr[tid], sJTr[tid]);
}


// Host-side LMA update (solve (JᵗJ + λI) δ = Jᵗr)
float update_parameters(float* JTJ, float* JTr, float* new_params, const float * params, const float * y, float lambda) {
  // Solve PxP linear system using basic Gaussian elimination (naive, for demo)
  float A[P][5]; // PxP + rhs
  for (int i = 0; i < P; ++i)
    for (int j = 0; j < P; ++j)
      A[i][j] = JTJ[i * P + j] + (i == j ? lambda : 0.0f);
  for (int i = 0; i < P; ++i)
    A[i][P] = JTr[i];


  //print the augmented matrix A
  // printf("Augmented matrix A:\n");
  // for (int i = 0; i < P; ++i) {
  //   for (int j = 0; j < 5; ++j) {
  //     printf("%8.Pf ", A[i][j]);
  //   }
  //   printf("\n");
  // }

  // Gaussian elimination with partial pivoting
  for (int i = 0; i < P; ++i) {
    // Partial pivoting: find the row with the largest absolute value in column i
    int maxRow = i;
    float maxVal = fabsf(A[i][i]);
    for (int k = i + 1; k < P; ++k) {
      if (fabsf(A[k][i]) > maxVal) {
        maxVal = fabsf(A[k][i]);
        maxRow = k;
      }
    }
    // Swap rows if needed
    if (maxRow != i) {
      for (int j = 0; j <= P; ++j) {
        float tmp = A[i][j];
        A[i][j] = A[maxRow][j];
        A[maxRow][j] = tmp;
      }
    }
    float pivot = A[i][i];
    if (fabsf(pivot) < 1e-8f) pivot = 1e-8f; // Avoid division by zero
    for (int j = 0; j <= P; ++j)
      A[i][j] /= pivot;
    for (int k = 0; k < P; ++k) {
      if (k == i) continue;
      float factor = A[k][i];
      for (int j = 0; j <= P; ++j)
        A[k][j] -= factor * A[i][j];
    }
  }

  // Update parameters
  for (int i = 0; i < P; ++i) {
    new_params[i] = params[i] + A[i][P];
    // printf("Updated parameter %d: %.4f <- %.4f + %.4f\n", i, new_params[i], params[i], A[i][4]);
  }

  // calculate SSR
  float SSR = 0.0f;
  for (int i = 0; i < N; ++i) {
      float fi = model(i , new_params[0], new_params[1], new_params[2], new_params[3]);
      float ri = (y[i]  - fi);
      SSR += ri * ri;
  }
  return SSR;
    
}

int main() {
    float *y, *d_y, *d_res, *d_J, *d_JTJ, *d_JTr, *d_SSR;
    float params[P] = { 5.0f, 5.0f, 1.0f, 0.5f }; // Initial guess: A, T, R, B
    float new_params[P];

    float noise_level = 0.0f;

    y = new float[N];

    // Generate synthetic data
    for (int i = 0; i < N; ++i) {
      float true_val  = model(i, 10.0f, 503.0f, 80.0f, 1.0f);
      y[i] = true_val + noise_level * ((rand() % 1000) / 1000.0f - 0.5f);
    }

    unsigned int time0 = getTime_us();

    // Allocate GPU memory
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_res, N * sizeof(float));
    hipMalloc(&d_SSR, sizeof(float));
    hipMalloc(&d_J, N * P * sizeof(float));
    hipMalloc(&d_JTJ, P * P * sizeof(float));
    hipMalloc(&d_JTr, P * sizeof(float));

    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    float lambda = 1.0f;

    unsigned int time1 = getTime_us();
    printf("Memory allocation and data transfer took %u us\n", time1 - time0 );

    int iter = 0;

    float JTJ[P*P], JTr[P], SSR;    
    do{
      // unsigned int time2 = getTime_us();

      // printf("----------------------- Iteration %d -----------------------\n", iter);
      // printf("Current parameters: A = %.4f, T = %.4f, R = %.4f, B = %.4f\n",
            // params[0], params[1], params[2], params[3]);
      
      hipMemset(d_SSR, 0, sizeof(float));
      compute_residuals_SSR_and_jacobian<<<BLOCKS, THREADS>>>(d_y, d_res, d_SSR, d_J, params[0], params[1], params[2], params[3]);
      hipDeviceSynchronize();

      hipMemset(d_JTJ, 0, sizeof(float) * P * P);
      hipMemset(d_JTr, 0, sizeof(float) * P);

      compute_JTJ_JTr<<<BLOCKS, THREADS>>>(d_J, d_res, d_JTJ, d_JTr);
      hipDeviceSynchronize();

      // unsigned int time2a = getTime_us();
      // printf("Iteration %d: Kernel execution took %u us\n", iter, time2a - time2);

      hipMemcpy( JTJ, d_JTJ, sizeof(float) * P * P, hipMemcpyDeviceToHost);
      hipMemcpy( JTr, d_JTr, sizeof(float) * P, hipMemcpyDeviceToHost);
      hipMemcpy(&SSR, d_SSR, sizeof(float), hipMemcpyDeviceToHost);

      // unsigned int time2b = getTime_us();
      // printf("Iteration %d: Data transfer took %u us\n", iter, time2b - time2a);

      float new_SSR = update_parameters(JTJ, JTr, new_params, params, y, lambda);
      // unsigned int time3 = getTime_us();
      // printf("Iteration %d: Time for kernel execution and parameter update: %u us\n", iter, time3 - time2);
      
      if( fabs(new_SSR - SSR) < 1e-6f ) break;

      if (new_SSR < SSR) {
        lambda *= 0.1f;
        // printf("$$$$$$$$$ Accepting new parameters.\n");
        for (int i = 0; i < P; ++i) params[i] = new_params[i];
        SSR = new_SSR;
      } else {
        lambda *= 10.0f;
      }

      // unsigned int time4 = getTime_us();
      // printf("Iteration %d: A = %.4f, T = %.4f, R = %.4f, B = %.4f | SSR: %f, new SSR: %f | lambda : %.3e\n",
      //       iter, new_params[0], new_params[1], new_params[2], new_params[3], SSR, new_SSR, lambda);

      
    }while(iter++ < 200 && 1e+12 > lambda && lambda > 1e-12);

    double var = SSR / dF; // variance
    double error[P];      // standard deviation
    
    float JTJ_inv[P*P];
    // Invert JTJ matrix
    // Invert 4x4 matrix JTJ using Gauss-Jordan elimination
    for (int i = 0; i < P * P; ++i) JTJ_inv[i] = (i % (P + 1)) ? 0.0f : 1.0f; // Identity matrix
    float temp[P * P];
    for (int i = 0; i < P * P; ++i) temp[i] = JTJ[i];
    // Gauss-Jordan elimination
    for (int col = 0; col < P; ++col) {
      float pivot = temp[col * P + col];
      if (fabsf(pivot) < 1e-8f) pivot = 1e-8f;
      for (int j = 0; j < P; ++j) {
        temp[col * P + j] /= pivot;
        JTJ_inv[col * P + j] /= pivot;
      }
      for (int row = 0; row < P; ++row) {
        if (row == col) continue;
        float factor = temp[row * P + col];
        for (int j = 0; j < P; ++j) {
          temp[row * P + j] -= factor * temp[col * P + j];
          JTJ_inv[row * P + j] -= factor * JTJ_inv[col * P + j];
        }
      }
    }
      
    for (int i = 0; i < P; ++i) error[i] = sqrt(var * JTJ[i * P + i]);

    unsigned int time4 = getTime_us();
    printf("Total execution time: %u us\n", time4 - time0);
    
    printf("==================== Fitting result: \n");
    printf("Number of iteraciton: %d, SSR = %f\n", iter, SSR);
    for (int i = 0; i < P; ++i) {
    printf("par[%d] = %12.6f (%.6f)\n", i, params[i], error[i] );
    }
    printf("######################################## end of LMA\n");

    // Cleanup
    hipFree(d_y); hipFree(d_res); hipFree(d_J);
    hipFree(d_JTJ); hipFree(d_JTr);
    delete[] y;

    return 0;
}
