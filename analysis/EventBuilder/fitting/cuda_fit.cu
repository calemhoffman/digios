#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>

#include "Matrix.h" // Simple matrix class for small matrices
#include "timer.h"

// to compile nvcc cuda_fit.cu -o cuda_fit -lcublas


#include <hipblas.h>


__global__ void computeYf(float* dx, float *dY, float* dYf, const float *para, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float A = para[0];
    float T = para[1];
    float R = para[2];
    float B = para[3];
    if (idx < n) {
        // Load parameters from device memory
        float model = A / (1.0f + expf((dx[idx] - T) / R)) + B;
        dYf[idx] = dY[idx] - model; // Y - f(p_0)
    }
}

__global__ void computeJacobian(float* dx, float* dJ, const float * para, int n, int p) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float A = para[0];
    float T = para[1];
    float R = para[2];
    if (idx < n) {
        float exp_term = expf((dx[idx] - T) / R);
        float denom = 1.0f + exp_term;
        // Fill the Jacobian matrix J
        if( exp_term < 1e+20 ) {
            dJ[idx * p + 0] = 1.0f / denom; // dA
            dJ[idx * p + 1] = A * exp_term / (R * denom * denom); // dT
            dJ[idx * p + 2] = (A * (dx[idx] - T) * exp_term) / (R * R * denom * denom); // dR
        }else{
            dJ[idx * p + 0] = 0.0f; // dA
            dJ[idx * p + 1] = 0.0f; // dT
            dJ[idx * p + 2] = 0.0f; // dR
        }
        dJ[idx * p + 3] = 1.0f; // dB
    }
}


void NonLinearRegression(double tolerance = 1e-5, int max_iter = 10000, double lambdaUp = 10, double lambdaDown = 10){
    
    // Simulated data
    int n = 500;
    std::vector<float> x(n);
    std::vector<float> y(n);
    std::vector<float> para = {12.0f, 100.0f, 5.0f, 7.0f}; // Initial guess for A, T, R, B
    int p = para.size();
    
    const int dF = n - p; // degrees of freedom
    
    float maxNoise = 0.0f; // Maximum noise level
    for( int i = 0; i < n; i++ ) {
        x[i] = i;
        y[i] = 10.0f / (1 + expf((i - 200.0f) / 3.0f)) + 8.0f + (rand() % 100) / 100.0 * maxNoise; // Adding some noise
        y[i] = round(y[i] * 1000.0f) / 1000.0f; // Round to 3 decimal places
        // printf("{%.0f, %.3f},", x[i], y[i]);
    }
    // printf("\n");
    
    // cuda memory pointers
    float *dx, *dy;
    hipMalloc(&dx, n * sizeof(float));
    hipMalloc(&dy, n * sizeof(float));
    float *dYf, *dJ;
    hipMalloc(&dYf, n * sizeof(float)); // Yf = Y - f(p_0)
    hipMalloc(&dJ, n * p * sizeof(float)); // Jacobian matrix
    float * dpara_new;
    hipMalloc(&dpara_new, p * sizeof(float)); // New parameters
    float *dH;
    hipMalloc(&dH, p * p * sizeof(float)); // Hessian matrix
    float *dG;
    hipMalloc(&dG, p * sizeof(float)); // Gradient vector

    hipMemcpy(dx, x.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;


    float lambda = 1.0f; // LMA factor
    float SSR = 0.0f;
    int count = 0; // count of iterations

    float new_SSR = 0.0f;

    Matrix H_inv(p, p); // Inverse of Hessian matrix;

    // Compute predicted values
    hipMemset(dYf, 0, n * sizeof(float)); // Initialize dYf to zero
    computeYf<<<blocks, threadsPerBlock>>>(dx, dy, dYf, para.data(), n);
    hipDeviceSynchronize(); // ensure kernel execution is complete

    
    // // Compute the SSR (mean squared error) in host
    std::vector<float> Yf(n);
    hipMemcpy(Yf.data(), dYf, n * sizeof(float), hipMemcpyDeviceToHost);
    SSR = 0.0f;
    for (int i = 0; i < n; i++){
        SSR += Yf[i] * Yf[i];
        // printf(" %d | %f - %f = %f \n", i, y[i], (para[0] / (1.0f + expf((x[i] - para[1]) / para[2])) + para[3]), Yf[i]);
    }
    printf("Initial SSR = %f\n", SSR);
    

    float deltaSSR = 0.f;

    //^=========================================

    do{

        // print current parameters
        printf("Iteration %d: SSR = %f,  {%f, %f, %f, %f}, Lambda = %.3e\n", count, SSR, para[0], para[1], para[2], para[3], lambda);

        // Compute Jacobian matrix J
        computeJacobian<<<blocks, threadsPerBlock>>>(dx, dJ, para.data(), n, p);
        hipDeviceSynchronize(); 

        // CUDA matrix multiplication for H = J^T * J
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        const float alpha = 1.0f; // Scaling factor for the matrix multiplication
        const float beta = 0.0f; 
                
        // dJ is stored as a n x p row-major matrix. CUBLAS expects column-major format,
        // so it sees dJ as a p x n matrix, which is equivalent to J^T.
        // To compute H = J^T * J, we can do (J^T) * (J^T)^T.
        // This corresponds to hipblasSgemm with opA = HIPBLAS_OP_N and opB = HIPBLAS_OP_T.
        hipblasSgemm(handle,
            HIPBLAS_OP_N, // op(A) = A = J^T
            HIPBLAS_OP_T, // op(B) = A^T = (J^T)^T = J
            p,           // m = rows of op(A) and H
            p,           // n = columns of op(B) and H
            n,           // k = inner dimension
            &alpha,
            dJ, p,       // A is p x n, lda = p
            dJ, p,       // B is p x n, ldb = p
            &beta,
            dH, p);      // C is p x p, ldc = p
        
        hipblasDestroy(handle);

        // copy dH back to Host, since H is small matrix, 4x4
        std::vector<float> H_matrix(p * p);
        hipMemcpy(H_matrix.data(), dH, p * p * sizeof(float), hipMemcpyDeviceToHost);

        // Compute gradient: g = J^T * Yf 
        hipblasHandle_t handle3;
        hipblasCreate(&handle3);
        // To compute g = J^T * Yf, we can use the fact that dJ is already J^T in column-major format.
        // So we can use hipblasSgemv with opA = HIPBLAS_OP_N.
        hipblasSgemv(handle3, HIPBLAS_OP_N, p, n, &alpha, dJ, p, dYf, 1, &beta, dG, 1); // g = J^T * Yf
        hipblasDestroy(handle3);

        // copy dG back to Host
        std::vector<float> G_vector(p);
        hipMemcpy(G_vector.data(), dG, p * sizeof(float), hipMemcpyDeviceToHost);

        // Convert H and G to Matrix class
        Matrix H(p, p);
        for (int i = 0; i < p; i++) {
            for (int j = 0; j < p; j++) {
                H(i, j) = H_matrix[i * p + j] + (i == j ? lambda : 0.0f); // Add lambda to diagonal elements
            }
        }
        Matrix G(p, 1);
        for (int i = 0; i < p; i++) {
            G(i, 0) = G_vector[i]; 
        }

        // Compute inverse of H using Matrix class
        if (std::isnan(Det(H)) || Det(H) == 0.0) {
            std::cout << "Hessian is singular or NaN, cannot compute inverse." << std::endl;
            break;
        }

        H_inv = Inv(H);

        // Update parameters: p = p - H_inv * G
        Matrix delta = H_inv * G; // delta = H_inv * G

        std::vector<float> para_new = para; // Copy current parameters
        for (int i = 0; i < p; i++) para_new[i] += delta(i, 0); // Update parameters    
        
        // Calculate the new SSR.
        // Compute predicted values
        hipMemcpy(dpara_new, para_new.data(), p * sizeof(float), hipMemcpyHostToDevice);

        computeYf<<<blocks, threadsPerBlock>>>(dx, dy, dYf, dpara_new, n);
        hipDeviceSynchronize(); // ensure kernel execution is complete
        
        // Compute the SSR (mean squared error)
        new_SSR = 0.0f;
        hipMemcpy(Yf.data(), dYf, n * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0; i < n; i++) new_SSR += Yf[i] * Yf[i];

        // printf("         new SSR = %f, parameters: [%f, %f, %f, %f]\n", new_SSR, para_new[0], para_new[1], para_new[2], para_new[3]);
        deltaSSR = fabs(SSR - new_SSR);

        if(  new_SSR < SSR ) {
            lambda /= lambdaDown; // Increase lambda, leaning towards Newton's method
            para = para_new; // Update parameters
            SSR = new_SSR; // Update SSR
            if( deltaSSR < tolerance ) break;

        } else {
            lambda *= lambdaUp; // Increase lambda
        }

        count++;

        // if( count % 10 == 0) {
            // printf("Iteration %d: SSR = %f, Parameters = [%f, %f, %f, %f], Lambda = %f\n", count, SSR, para[0], para[1], para[2], para[3], lambda);
        // }

    }while( count < 1000 && 1e+12 > lambda && lambda > 1e-12);

    
    if( count >= max_iter ){
      printf("Warning: LMA did not converge within the maximum number of iterations (%d)\n", max_iter);
    }

    if( deltaSSR <= tolerance * SSR ){
      printf("LMA converged after %d iterations with SSR = %f\n", count, SSR);
    }else{
      printf("LMA did not converge, last SSR = %f after %d iterations\n", SSR, count); 
    }

    //calculate errors;
    double var = SSR / dF; // variance
    std::vector<float> error(p, 0.0f);
    for (int i = 0; i < p; ++i)  error[i] = sqrt(var * H_inv(i, i)); // standard error

    // print the final parameters
    printf("==================== Fitting result: \n");
    printf("SSR = %f\n", SSR);
    for (int i = 0; i < p; ++i) {
      printf("par[%d] = %f (%f)\n", i, para[i], error[i] );
    }
    printf("######################################## end of LMA\n");


    hipFree(dx);
    hipFree(dy);
    hipFree(dYf);
    hipFree(dJ);
    hipFree(dH);
    hipFree(dG);
    hipFree(dpara_new);

    return;

}

int main() {

    NonLinearRegression();

    // // Data points
    // std::vector<float> y = {
    //     15004, 15000, 14997, 14994, 14994, 14995, 14994, 14992, 14988, 14983, 14987, 14993, 14990, 14989, 14989, 14991, 14993, 15000, 15001, 15000, 15001, 15004, 14999, 14996, 14991, 14992, 14986, 14983, 14984, 14980, 14979, 14979, 14981, 14984, 14984, 14986, 14986, 14987, 14988, 14981, 14980, 14980, 14976, 14976, 14978, 14969, 14964, 14956, 14954, 14950, 14951, 14947, 14951, 14956, 14952, 14954, 14959, 14958, 14954, 14950, 14954, 14958, 14953, 14945, 14942, 14942, 14938, 14937, 14937, 14935, 14935, 14927, 14927, 14931, 14926, 14926, 14926, 14930, 14930, 14925, 14921, 14916, 14912, 14907, 14904, 14902, 14897, 14887, 14873, 14865, 14841, 14827, 14811, 14785, 14764, 14744, 14723, 14702, 14682, 14682, 14645, 14621, 14600, 14578, 14556, 14539, 14509, 14494, 14476, 14461, 14440, 14424, 14409, 14395, 14383, 14372, 14363, 14361, 14353, 14348, 14340, 14335, 14332, 14325, 14321, 14314, 14305, 14305, 14301, 14301, 14297, 14297, 14293, 14292, 14289, 14282, 14279, 14268, 14260, 14255, 14254, 14251, 14249, 14242, 14236, 14234, 14227, 14228, 14223, 14220, 14221, 14216, 14210, 14206, 14203, 14196, 14193, 14186, 14185, 14184, 14181, 14179, 14174, 14169, 14164, 14157, 14158, 14148, 14141, 14139, 14137, 14136, 14131, 14127, 14122, 14119, 14116, 14109, 14104, 14103, 14100, 14088, 14080, 14074, 14071, 14066, 14064, 14058, 14059, 14062, 14058, 14052, 14050, 14046, 14040, 14035, 14033, 14026, 14022, 14012, 14006, 14001, 14004, 13997, 13997, 13996, 13996, 13996, 13985, 13980, 13971, 13967, 13959, 13953, 13952, 13952, 13946, 13940, 13936, 13930, 13925, 13915, 13914, 13908, 13907, 13902, 13896, 13895, 13893, 13891, 13889, 13886, 13876, 13876, 13872, 13867, 13857, 13853, 13853, 13845, 13843, 13843, 13843, 13827, 13819, 13816, 13817, 13810, 13810, 13801, 13792, 13786, 13780, 13776, 13768, 13761, 13754, 13749, 13743, 13737, 13729, 13724, 13720, 13720, 13716, 13715, 13716, 13709, 13707, 13706, 13704, 13702, 13696, 13687, 13686, 13677, 13673, 13669, 13665, 13663, 13661, 13660, 13659, 13658, 13652, 13647, 13644, 13641, 13636, 13640, 13636, 13635, 13635, 13634, 13627, 13622, 13617, 13606, 13605, 13594, 13586, 13584, 13583, 13578, 13574, 13568, 13565, 13563, 13557, 13557, 13551, 13541, 13539, 13530, 13521, 13508, 13497, 13490, 13480, 13471, 13462, 13458, 13454, 13448, 13443, 13439, 13438, 13431, 13431, 13426, 13420, 13414, 13410, 13409, 13408, 13397, 13389, 13390, 13390, 13390, 13392, 13391, 13392, 13387, 13387, 13382, 13378, 13375, 13374, 13375, 13369, 13367, 13368, 13364, 13367, 13361, 13359, 13353, 13347, 13339, 13338, 13332, 13327, 13323, 13318, 13313, 13309, 13304, 13305, 13302, 13304, 13298, 13296, 13289, 13277, 13273, 13265, 13254, 13244, 13238, 13237, 13232, 13225, 13219, 13220, 13214, 13202, 13196, 13190, 13188, 13182, 13175, 13168, 13164, 13163, 13152, 13146, 13142, 13136, 13133, 13126, 13119, 13115, 13105, 13104, 13098, 13097, 13092, 13086, 13078, 13070, 13062, 13053, 13038, 13032, 13028, 13024, 13022, 13020, 13019, 13019, 13015, 13019, 13019, 13020, 13025, 13027, 13031, 13032, 13032, 13030, 13028, 13025, 13026, 13021, 13014, 13012, 13006, 13004, 12996, 12993, 12992, 12991, 12989, 12993, 12997, 13004, 13003, 13009, 13010, 13007, 13007, 13005, 13005, 13000, 13000, 13000, 12994, 12995, 12996, 12996, 12999, 13001, 12997, 12995, 12995, 12994, 12998, 13000, 12999, 13000, 13001, 12999, 12997, 12995, 12996, 12992, 12983, 12974, 12972, 12978, 12976, 12977, 12980, 12985, 12984, 12977, 12974, 12979, 12978, 12980, 12973, 12977, 12974, 12969, 12970, 12973, 12970, 12961, 12963, 12965, 12963, 12957, 12961, 12958, 12961, 12959, 12963, 12962, 12968, 12963, 12965, 12962, 12963, 12963, 12965, 12966, 12966, 12971, 12971, 12969, 12968, 12970, 12968, 12962, 12961, 12955, 12954, 12948, 12948, 12951, 12953, 12958, 12954, 12954, 12953, 12958, 12958, 12954, 12955, 12952, 12947, 12949, 12948, 12947, 12949, 12947, 12949, 12949, 12947, 12947, 12942, 12940, 12938, 12940, 12940, 12938, 12938, 12939, 12937, 12934, 12933, 12936, 12937, 12933, 12933, 12933, 12929, 12925, 12924, 12922, 12927, 12930, 12935, 12935, 12934, 12935, 12930, 12932, 12927, 12927, 12926, 12927, 12929, 12930, 12934, 12936, 12936, 12934, 12932, 12931, 12926, 12926, 12933, 12928, 12923, 12924, 12921, 12918, 12915, 12913, 12908, 12909, 12909, 12911, 12917, 12920, 12922, 12922, 12920, 12915, 12919, 12919, 12919, 12920, 12920, 12922, 12918, 12918, 12919, 12916, 12918, 12912, 12909, 12917, 12911, 12908, 12911, 12914, 12920, 12920, 12916, 12914, 12917, 12914, 12909, 12908, 12907, 12911, 12913, 12913, 12914, 12912, 12909, 12908, 12905, 12900, 12904, 12908, 12913, 12916, 12918, 12918, 12924, 12930, 12932, 12927, 12930, 12928, 12921, 12910, 12910, 12911, 12909, 12911, 12908, 12916, 12916, 12914, 12919, 12920, 12927, 12926, 12928, 12931, 12928, 12925, 12928, 12923, 12923, 12919, 12917, 12916, 12916, 12918, 12916, 12913, 12916, 12911, 12919, 12922, 12920, 12926, 12928, 12930, 12930, 12935, 12933, 12934, 12930, 12932, 12932, 12926, 12922, 12922, 12921, 12922, 12918, 12914, 12914, 12913, 12916, 12915, 12915, 12919, 12915, 12914, 12912, 12912, 12907, 12905, 12902, 12901, 12898, 12904, 12901, 12902, 12901, 12908, 12909, 12907, 12910, 12912, 12913, 12911, 12910, 12911, 12912, 12910, 12916, 12913, 12914, 12914, 12915, 12914, 12917, 12915, 12913, 12911, 12907, 12902, 12907, 12905, 12909, 12907, 12905, 12904, 12904, 12902, 12905, 12903, 12907, 12910, 12914, 12911, 12912, 12910, 12909, 12910, 12916, 12916, 12919, 12921, 12923, 12925, 12922, 12920, 12919, 12916, 12913, 12914, 12922, 12925, 12924, 12922, 12923, 12916, 12914, 12909, 12906, 12902, 12908, 12907, 12906, 12907, 12912, 12912, 12915, 12918, 12920, 12924, 12922, 12920, 12922, 12914, 12913, 12909, 12911, 12914, 12914, 12910, 12907, 12907, 12907, 12909, 12913, 12911, 12909, 12905, 12908, 12907, 12907, 12908, 12906, 12902, 12903, 12901, 12898, 12896, 12894, 12895, 12896, 12899, 12894, 12892, 12891, 12885, 12889, 12897, 12896, 12901, 12902, 12901, 12899, 12902, 12900, 12901, 12899, 12901, 12899, 12895, 12894, 12897, 12898, 12901, 12904, 12908, 12910, 12905, 12907, 12903, 12898, 12901, 12904, 12906, 12909, 12911, 12909, 12908, 12913, 12913, 12913, 12913, 12913, 12913, 12915, 12912, 12910, 12909, 12901, 12900, 12905, 12906, 12909, 12912, 12917, 12922, 12924, 12927, 12918, 12912, 12911, 12906, 12904, 12903, 12902, 12904, 12905, 12909, 12909, 12910, 12915, 12910, 12910, 12910, 12910
    // };

    // std::vector<float> x;
    // for( int i = 0; i < y.size(); i++ ) x.push_back(i);

    return 0;
}