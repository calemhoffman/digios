#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>

// to compile nvcc cuda_fit.cu -o cuda_fit

// CUDA kernel to compute the model: A/(1+exp(-(x-T)/R)) + B
__global__ void computeModel(float* x, float* y_pred, float A, float T, float R, float B, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y_pred[idx] = A / (1.0f + expf((x[idx] - T) / R)) + B;
    }
}

// CUDA kernel to compute the loss (mean squared error)
__global__ void computeLoss(float* y_true, float* y_pred, float* loss, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float diff = y_true[idx] - y_pred[idx];
        atomicAdd(loss, diff * diff);
    }
}

// CUDA kernel to compute gradients for A, T, R, B
__global__ void computeGradients(float* x, float* y_true, float* y_pred, float* gradA, float* gradT, float* gradR, float* gradB, float A, float T, float R, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float exp_term = expf((x[idx] - T) / R);
        float denom = 1.0f + exp_term;
        float diff = y_pred[idx] - y_true[idx];

        // Partial derivatives (chain rule applied)
        float dA = 1.0f / denom;
        float dB = 1.0f;
        float dT = (A * exp_term) / (R * denom * denom);
        float dR = (A * (-T + x[idx]) * exp_term) / (R * R * denom * denom);

        atomicAdd(gradA, 2.0f * diff * dA);
        atomicAdd(gradT, 2.0f * diff * dT);
        atomicAdd(gradR, 2.0f * diff * dR);
        atomicAdd(gradB, 2.0f * diff * dB);
    }
}

// CUDA kernel to compute the diagonal of the Hessian (second derivatives)
__global__ void computeHessianDiag(float* x, float* y_true, float* y_pred, float* hessA, float* hessT, float* hessR, float* hessB, float A, float T, float R, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float exp_term = expf((x[idx] - T) / R);
        float denom = 1.0f + exp_term;
        float denom2 = denom * denom;
        float diff = y_pred[idx] - y_true[idx];

        // Second derivatives (diagonal only)
        float dA = 1.0f / denom;
        float dT = (A * exp_term) / (R * denom2);
        float dR = (A * (x[idx] - T) * exp_term) / (R * R * denom2);
        float dB = 1.0f;

        atomicAdd(hessA, 2.0f * dA * dA);
        atomicAdd(hessT, 2.0f * dT * dT);
        atomicAdd(hessR, 2.0f * dR * dR);
        atomicAdd(hessB, 2.0f * dB * dB);
    }
}

void computeUncertainties(std::vector<float>& x, std::vector<float>& y, float A, float T, float R, float B, float& sigmaA, float& sigmaT, float& sigmaR, float& sigmaB) {
    int n = x.size();
    float *d_x, *d_y, *d_y_pred, *d_hessA, *d_hessT, *d_hessR, *d_hessB;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_y_pred, n * sizeof(float));
    hipMalloc(&d_hessA, sizeof(float));
    hipMalloc(&d_hessT, sizeof(float));
    hipMalloc(&d_hessR, sizeof(float));
    hipMalloc(&d_hessB, sizeof(float));

    hipMemcpy(d_x, x.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Compute model predictions
    computeModel<<<blocks, threadsPerBlock>>>(d_x, d_y_pred, A, T, R, B, n);
    hipDeviceSynchronize();

    // Initialize Hessian diagonals
    hipMemset(d_hessA, 0, sizeof(float));
    hipMemset(d_hessT, 0, sizeof(float));
    hipMemset(d_hessR, 0, sizeof(float));
    hipMemset(d_hessB, 0, sizeof(float));

    // Compute Hessian diagonal
    computeHessianDiag<<<blocks, threadsPerBlock>>>(d_x, d_y, d_y_pred, d_hessA, d_hessT, d_hessR, d_hessB, A, T, R, n);
    hipDeviceSynchronize();

    float hessA = 0.0f, hessT = 0.0f, hessR = 0.0f, hessB = 0.0f;
    hipMemcpy(&hessA, d_hessA, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&hessT, d_hessT, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&hessR, d_hessR, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&hessB, d_hessB, sizeof(float), hipMemcpyDeviceToHost);

    // Uncertainty estimation: sqrt(1 / Hessian)
    sigmaA = sqrtf(1.0f / hessA);
    sigmaT = sqrtf(1.0f / hessT);
    sigmaR = sqrtf(1.0f / hessR);
    sigmaB = sqrtf(1.0f / hessB);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_y_pred);
    hipFree(d_hessA);
    hipFree(d_hessT);
    hipFree(d_hessR);
    hipFree(d_hessB);
}

void fitCurve(std::vector<float>& x, std::vector<float>& y, float& A, float& T, float& R, float& B, float initial_lr = 0.2f) {
    int n = x.size();

    // Device memory
    float *d_x, *d_y, *d_y_pred, *d_loss, *d_gradA, *d_gradT, *d_gradR, *d_gradB;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_y_pred, n * sizeof(float));
    hipMalloc(&d_loss, sizeof(float));
    hipMalloc(&d_gradA, sizeof(float));
    hipMalloc(&d_gradT, sizeof(float));
    hipMalloc(&d_gradR, sizeof(float));
    hipMalloc(&d_gradB, sizeof(float));

    // Copy input data to device
    hipMemcpy(d_x, x.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), n * sizeof(float), hipMemcpyHostToDevice);

    // CUDA grid configuration
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Levenberg-Marquardt parameters
    float lambda = 1.0f; // damping factor
    float lambda_up = 5.0f;
    float lambda_down = 0.5f;
    float learning_rate = initial_lr;

    int nIterations = 0;
    double tolerance = 1e-3;
    float loss = 0.0f, old_loss = 0.0;
    float tolerance_check = 0.0f;
    
    do {
        // Initialize loss and gradients
        float gradA = 0.0f, gradT = 0.0f, gradR = 0.0f, gradB = 0.0f;
        hipMemset(d_loss, 0, sizeof(float));
        hipMemset(d_gradA, 0, sizeof(float));
        hipMemset(d_gradT, 0, sizeof(float));
        hipMemset(d_gradR, 0, sizeof(float));
        hipMemset(d_gradB, 0, sizeof(float));

        // Compute model predictions
        computeModel<<<blocks, threadsPerBlock>>>(d_x, d_y_pred, A, T, R, B, n);
        hipDeviceSynchronize();

        // Compute loss
        computeLoss<<<blocks, threadsPerBlock>>>(d_y, d_y_pred, d_loss, n);
        hipDeviceSynchronize();
        hipMemcpy(&loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
        loss /= n;

        // Compute gradients
        computeGradients<<<blocks, threadsPerBlock>>>(d_x, d_y, d_y_pred, d_gradA, d_gradT, d_gradR, d_gradB, A, T, R, n);
        hipDeviceSynchronize();
        hipMemcpy(&gradA, d_gradA, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&gradT, d_gradT, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&gradR, d_gradR, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&gradB, d_gradB, sizeof(float), hipMemcpyDeviceToHost);

        // Save current parameters
        float A_old = A, T_old = T, R_old = R, B_old = B;

        // LM update: add damping to gradient step
        A -= learning_rate * gradA / (n + lambda);
        T -= learning_rate * gradT / (n + lambda);
        R -= learning_rate * gradR / (n + lambda);
        B -= learning_rate * gradB / (n + lambda);

        // Recompute loss with new parameters
        hipMemset(d_loss, 0, sizeof(float));
        computeModel<<<blocks, threadsPerBlock>>>(d_x, d_y_pred, A, T, R, B, n);
        hipDeviceSynchronize();
        computeLoss<<<blocks, threadsPerBlock>>>(d_y, d_y_pred, d_loss, n);
        hipDeviceSynchronize();
        float new_loss = 0.0f;
        hipMemcpy(&new_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
        new_loss /= n;

        // Adaptive lambda adjustment
        if (new_loss < loss) {
            // Accept update, decrease lambda
            lambda *= lambda_down;
            old_loss = new_loss;
        } else {
            // Reject update, increase lambda and revert parameters
            lambda *= lambda_up;
            A = A_old;
            T = T_old;
            R = R_old;
            B = B_old;
            continue;
        }

        tolerance_check = fabs(loss - old_loss)/ fabs(old_loss);

        if (nIterations % 10 == 0){
            printf(" %f, %f, %f|%f| A: %f, T: %f, R: %f, B: %f, lambda: %f\n", old_loss, loss, new_loss, tolerance_check, A, T, R, B, lambda);
        }

        if (nIterations > 50 && tolerance_check < tolerance) {
            printf("Convergence reached. num of iteration : %d | loss tolerance %f\n", nIterations, tolerance_check);
            break;
        }

    } while ( nIterations++ < 10000);

    //if not converged, print warning
    if (nIterations >= 10000) {
        printf("Warning: Maximum iterations reached without convergence. tolerance %f\n", tolerance_check);
    }

    // Clean up
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_y_pred);
    hipFree(d_loss);
    hipFree(d_gradA);
    hipFree(d_gradT);
    hipFree(d_gradR);
    hipFree(d_gradB);
}

int main() {

    // Data points
    std::vector<float> y = {
        15004, 15000, 14997, 14994, 14994, 14995, 14994, 14992, 14988, 14983, 14987, 14993, 14990, 14989, 14989, 14991, 14993, 15000, 15001, 15000, 15001, 15004, 14999, 14996, 14991, 14992, 14986, 14983, 14984, 14980, 14979, 14979, 14981, 14984, 14984, 14986, 14986, 14987, 14988, 14981, 14980, 14980, 14976, 14976, 14978, 14969, 14964, 14956, 14954, 14950, 14951, 14947, 14951, 14956, 14952, 14954, 14959, 14958, 14954, 14950, 14954, 14958, 14953, 14945, 14942, 14942, 14938, 14937, 14937, 14935, 14935, 14927, 14927, 14931, 14926, 14926, 14926, 14930, 14930, 14925, 14921, 14916, 14912, 14907, 14904, 14902, 14897, 14887, 14873, 14865, 14841, 14827, 14811, 14785, 14764, 14744, 14723, 14702, 14682, 14682, 14645, 14621, 14600, 14578, 14556, 14539, 14509, 14494, 14476, 14461, 14440, 14424, 14409, 14395, 14383, 14372, 14363, 14361, 14353, 14348, 14340, 14335, 14332, 14325, 14321, 14314, 14305, 14305, 14301, 14301, 14297, 14297, 14293, 14292, 14289, 14282, 14279, 14268, 14260, 14255, 14254, 14251, 14249, 14242, 14236, 14234, 14227, 14228, 14223, 14220, 14221, 14216, 14210, 14206, 14203, 14196, 14193, 14186, 14185, 14184, 14181, 14179, 14174, 14169, 14164, 14157, 14158, 14148, 14141, 14139, 14137, 14136, 14131, 14127, 14122, 14119, 14116, 14109, 14104, 14103, 14100, 14088, 14080, 14074, 14071, 14066, 14064, 14058, 14059, 14062, 14058, 14052, 14050, 14046, 14040, 14035, 14033, 14026, 14022, 14012, 14006, 14001, 14004, 13997, 13997, 13996, 13996, 13996, 13985, 13980, 13971, 13967, 13959, 13953, 13952, 13952, 13946, 13940, 13936, 13930, 13925, 13915, 13914, 13908, 13907, 13902, 13896, 13895, 13893, 13891, 13889, 13886, 13876, 13876, 13872, 13867, 13857, 13853, 13853, 13845, 13843, 13843, 13843, 13827, 13819, 13816, 13817, 13810, 13810, 13801, 13792, 13786, 13780, 13776, 13768, 13761, 13754, 13749, 13743, 13737, 13729, 13724, 13720, 13720, 13716, 13715, 13716, 13709, 13707, 13706, 13704, 13702, 13696, 13687, 13686, 13677, 13673, 13669, 13665, 13663, 13661, 13660, 13659, 13658, 13652, 13647, 13644, 13641, 13636, 13640, 13636, 13635, 13635, 13634, 13627, 13622, 13617, 13606, 13605, 13594, 13586, 13584, 13583, 13578, 13574, 13568, 13565, 13563, 13557, 13557, 13551, 13541, 13539, 13530, 13521, 13508, 13497, 13490, 13480, 13471, 13462, 13458, 13454, 13448, 13443, 13439, 13438, 13431, 13431, 13426, 13420, 13414, 13410, 13409, 13408, 13397, 13389, 13390, 13390, 13390, 13392, 13391, 13392, 13387, 13387, 13382, 13378, 13375, 13374, 13375, 13369, 13367, 13368, 13364, 13367, 13361, 13359, 13353, 13347, 13339, 13338, 13332, 13327, 13323, 13318, 13313, 13309, 13304, 13305, 13302, 13304, 13298, 13296, 13289, 13277, 13273, 13265, 13254, 13244, 13238, 13237, 13232, 13225, 13219, 13220, 13214, 13202, 13196, 13190, 13188, 13182, 13175, 13168, 13164, 13163, 13152, 13146, 13142, 13136, 13133, 13126, 13119, 13115, 13105, 13104, 13098, 13097, 13092, 13086, 13078, 13070, 13062, 13053, 13038, 13032, 13028, 13024, 13022, 13020, 13019, 13019, 13015, 13019, 13019, 13020, 13025, 13027, 13031, 13032, 13032, 13030, 13028, 13025, 13026, 13021, 13014, 13012, 13006, 13004, 12996, 12993, 12992, 12991, 12989, 12993, 12997, 13004, 13003, 13009, 13010, 13007, 13007, 13005, 13005, 13000, 13000, 13000, 12994, 12995, 12996, 12996, 12999, 13001, 12997, 12995, 12995, 12994, 12998, 13000, 12999, 13000, 13001, 12999, 12997, 12995, 12996, 12992, 12983, 12974, 12972, 12978, 12976, 12977, 12980, 12985, 12984, 12977, 12974, 12979, 12978, 12980, 12973, 12977, 12974, 12969, 12970, 12973, 12970, 12961, 12963, 12965, 12963, 12957, 12961, 12958, 12961, 12959, 12963, 12962, 12968, 12963, 12965, 12962, 12963, 12963, 12965, 12966, 12966, 12971, 12971, 12969, 12968, 12970, 12968, 12962, 12961, 12955, 12954, 12948, 12948, 12951, 12953, 12958, 12954, 12954, 12953, 12958, 12958, 12954, 12955, 12952, 12947, 12949, 12948, 12947, 12949, 12947, 12949, 12949, 12947, 12947, 12942, 12940, 12938, 12940, 12940, 12938, 12938, 12939, 12937, 12934, 12933, 12936, 12937, 12933, 12933, 12933, 12929, 12925, 12924, 12922, 12927, 12930, 12935, 12935, 12934, 12935, 12930, 12932, 12927, 12927, 12926, 12927, 12929, 12930, 12934, 12936, 12936, 12934, 12932, 12931, 12926, 12926, 12933, 12928, 12923, 12924, 12921, 12918, 12915, 12913, 12908, 12909, 12909, 12911, 12917, 12920, 12922, 12922, 12920, 12915, 12919, 12919, 12919, 12920, 12920, 12922, 12918, 12918, 12919, 12916, 12918, 12912, 12909, 12917, 12911, 12908, 12911, 12914, 12920, 12920, 12916, 12914, 12917, 12914, 12909, 12908, 12907, 12911, 12913, 12913, 12914, 12912, 12909, 12908, 12905, 12900, 12904, 12908, 12913, 12916, 12918, 12918, 12924, 12930, 12932, 12927, 12930, 12928, 12921, 12910, 12910, 12911, 12909, 12911, 12908, 12916, 12916, 12914, 12919, 12920, 12927, 12926, 12928, 12931, 12928, 12925, 12928, 12923, 12923, 12919, 12917, 12916, 12916, 12918, 12916, 12913, 12916, 12911, 12919, 12922, 12920, 12926, 12928, 12930, 12930, 12935, 12933, 12934, 12930, 12932, 12932, 12926, 12922, 12922, 12921, 12922, 12918, 12914, 12914, 12913, 12916, 12915, 12915, 12919, 12915, 12914, 12912, 12912, 12907, 12905, 12902, 12901, 12898, 12904, 12901, 12902, 12901, 12908, 12909, 12907, 12910, 12912, 12913, 12911, 12910, 12911, 12912, 12910, 12916, 12913, 12914, 12914, 12915, 12914, 12917, 12915, 12913, 12911, 12907, 12902, 12907, 12905, 12909, 12907, 12905, 12904, 12904, 12902, 12905, 12903, 12907, 12910, 12914, 12911, 12912, 12910, 12909, 12910, 12916, 12916, 12919, 12921, 12923, 12925, 12922, 12920, 12919, 12916, 12913, 12914, 12922, 12925, 12924, 12922, 12923, 12916, 12914, 12909, 12906, 12902, 12908, 12907, 12906, 12907, 12912, 12912, 12915, 12918, 12920, 12924, 12922, 12920, 12922, 12914, 12913, 12909, 12911, 12914, 12914, 12910, 12907, 12907, 12907, 12909, 12913, 12911, 12909, 12905, 12908, 12907, 12907, 12908, 12906, 12902, 12903, 12901, 12898, 12896, 12894, 12895, 12896, 12899, 12894, 12892, 12891, 12885, 12889, 12897, 12896, 12901, 12902, 12901, 12899, 12902, 12900, 12901, 12899, 12901, 12899, 12895, 12894, 12897, 12898, 12901, 12904, 12908, 12910, 12905, 12907, 12903, 12898, 12901, 12904, 12906, 12909, 12911, 12909, 12908, 12913, 12913, 12913, 12913, 12913, 12913, 12915, 12912, 12910, 12909, 12901, 12900, 12905, 12906, 12909, 12912, 12917, 12922, 12924, 12927, 12918, 12912, 12911, 12906, 12904, 12903, 12902, 12904, 12905, 12909, 12909, 12910, 12915, 12910, 12910, 12910, 12910
    };

    std::vector<float> x;
    for( int i = 0; i < y.size(); i++ ) x.push_back(i);

    // Initial parameter guesses
    float A = 3000.0, T = 160, R = 100.0, B = 12000;

    fitCurve(x, y, A, T, R, B);

    printf("Final parameters: (A, T, R, B) = %f, %f, %f, %f\n", A, T, R, B);
    // Compute uncertainties
    float sigmaA, sigmaT, sigmaR, sigmaB;
    computeUncertainties(x, y, A, T, R, B, sigmaA, sigmaT, sigmaR, sigmaB);
    printf("Uncertainties: (sigmaA, sigmaT, sigmaR, sigmaB) = %f, %f, %f, %f\n", sigmaA, sigmaT, sigmaR, sigmaB);

    return 0;
}